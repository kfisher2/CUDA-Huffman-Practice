
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <cstdlib>
// #include <helper_cuda.h>
// #include <helper_string.h>

/* Run with only HOST code *\

// Say goodbye to the universe
int main(void) {
	printf("Heat death boogaloo!\n");
	return 0;
}

*/


/* Run with DEVICEEEEEEE code */

__global__ void mykernel(void) { 
}

int main(void) {
  mykernel<<<1,1>>>();
  printf("HEat death boogaloo Universe!\n");
  return 0;
}

/*
blah blah comment not conflicting
__global__  indicates function that 1) runs on DEVICE , 2) called from the HOST

<<<   >>>   indicates function that    runs on DEVICE and is called from the HOST

*////////
