
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cstdio>

__global__ void add(int *a, int *b, int *c) {
  /*       BLOCKS        */
  // c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
  /*       VS THREADS    */
  c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

// RAWRRRRRRR COMMENT blah
void random_ints(int *a, int N) {
  for(int i = 0; i < N; ++i) {
    a[i] = rand();
  }
}

#define N 512
int main(void) {
  int *a, *b, *c;          // HOST copies of a, b, c
  int *d_a, *d_b, *d_c;    // DEVICE copies of a, b, c
  
  int size = N * sizeof(int);

  // Allocate space on DEVICE for DEVICE copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);
  // Allocate space on HOST for HOST copies of a, b, c
  a = (int *)malloc(size);
  b = (int *)malloc(size);
  c = (int *)malloc(size);

  // Set input values
  random_ints(a,N);
  random_ints(b,N);

  // Copy inputs to DEVICE
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  // Launch add() kernel on DEVICE
  /*  BLOCKS      */
  // add<<<N,1>>>(d_a, d_b, d_c);
  /*  VS THREADS  */
  add<<<1,N>>>(d_a, d_b, d_c);

  // Copy result back to HOST
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  // Test
  // would need loop now, too lazy
  // if(c != a + b) std::cout << "Incorrect addition with result: " << c << '\n';

  //Clean-up 
  free(a); free(b); free(c);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  return 0;

}

/* Moving to parallel

add<<<N, 1>>> we can execute add() N times in parallel

each parallel invocation is called a BLOCK
the collectino fo blocks is called a GRID

each invocation can refer to it's BLOCK index with
  blockIdx.x




*/
